#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <opencv2/opencv.hpp>
#include "gamma_correction.hpp"

using namespace cv;

__global__ void 
_gamma_correction_kernel_(const unsigned char* src, unsigned char* dst, const size_t rows, const size_t cols, const float gamma)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;
    const int id = idy * cols + idx;

    if(id >= rows * cols) {
        return;
    }

    const float inv_gamma = 1.0f / gamma;
    float value = round(pow(src[id] / 255.0f, inv_gamma) * 255.0f);
    value = (value > 255.0f) ? 255.0f : value;
    value = (value < 0) ? 0 : value;

    dst[id] = static_cast<uchar>(value);

}

void
_process_cuda(const unsigned char* src, unsigned char* dst, const size_t rows, const size_t cols, const float gamma)
{
    const size_t ARRAY_SIZE = rows * cols;
	const size_t ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned char);
    unsigned char* d_src;
    unsigned char* d_dst;
    hipMalloc((void**) &d_src, ARRAY_BYTES);
	hipMalloc((void**) &d_dst, ARRAY_BYTES);
    hipMemcpy(d_src, src, ARRAY_BYTES, hipMemcpyHostToDevice);

    const static int THREADS_COUNT = 8;
    dim3 threads(THREADS_COUNT, THREADS_COUNT);
    dim3 blocks(rows / threads.x + 1, cols / threads.y + 1);

    _gamma_correction_kernel_<<<blocks, threads>>>(d_src, d_dst, rows, cols, gamma);

    hipMemcpy(dst, d_dst, ARRAY_BYTES, hipMemcpyDeviceToHost);
    hipFree(d_src);
	hipFree(d_dst);
}

void
gamma_correction::_process(const cv::Mat& src, cv::Mat& dst, const float gamma)
{
    std::cout << "Run with CUDA" << std::endl;
    Mat bgr[3];
    Mat d_bgr[3];
    split(src, bgr);
    split(dst, d_bgr);
    _process_cuda(bgr[0].data, d_bgr[0].data, src.rows, src.cols, gamma);
    _process_cuda(bgr[1].data, d_bgr[1].data, src.rows, src.cols, gamma);
    _process_cuda(bgr[2].data, d_bgr[2].data, src.rows, src.cols, gamma);
    cv::merge(d_bgr, 3, dst);
}
